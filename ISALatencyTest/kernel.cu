#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#define PTX_ISA_VERSION_4_3									        // CHANGE THIS VALUE TO THE COMPUTE CAPABILITY OF YOUR CARD

#ifdef PTX_ISA_VERSION_4_3
#define PTX_ISA_VERSION_4_2
#endif

#ifdef PTX_ISA_VERSION_4_2
#define PTX_ISA_VERSION_4_1
#endif

#ifdef PTX_ISA_VERSION_4_1
#define PTX_ISA_VERSION_4_0
#endif

#ifdef PTX_ISA_VERSION_4_0
#define PTX_ISA_VERSION_3_2
#endif

#ifdef PTX_ISA_VERSION_3_2
#define PTX_ISA_VERSION_3_1
#endif

#ifdef PTX_ISA_VERSION_3_1
#define PTX_ISA_VERSION_3_0
#endif

#ifdef PTX_ISA_VERSION_3_1
#define PTX_ISA_VERSION_3_0
#endif

#ifdef PTX_ISA_VERSION_3_0
#define PTX_ISA_VERSION_2_2
#endif

#ifdef PTX_ISA_VERSION_2_3
#define PTX_ISA_VERSION_2_2
#endif

#ifdef PTX_ISA_VERSION_2_2
#define PTX_ISA_VERSION_2_1
#endif

#ifdef PTX_ISA_VERSION_2_1
#define PTX_ISA_VERSION_2_0
#endif

#ifdef PTX_ISA_VERSION_2_0
#define PTX_ISA_VERSION_1_x
#endif

typedef unsigned int(*TEST)(bool);

__device__ void Evaluate(TEST test)
{
	unsigned repetitionsleft = 5;
	unsigned lastlatency;
	unsigned newlatency;

	lastlatency = test(false);
	while (repetitionsleft) {
		newlatency = test(false);
		if (newlatency == lastlatency) repetitionsleft--;
		else {
			repetitionsleft = 5;
			lastlatency = newlatency;
		}
	}
	printf("%04u cyles for ", lastlatency);
	test(true);
	printf("\n");
}

#define NEXT_TEST_GROUP FirstTest

#include "test_bfind.h"
#include "test_brev.h"
#include "test_template.h"											// Add your test_INSTRUCTION.h file here

__device__ void NEXT_TEST_GROUP() {}

__global__ void LatencyTest()
{
	FirstTest();
}

int main()
{
	hipError_t cudaStatus = hipSuccess;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

	LatencyTest <<<1, 1>>>();

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

Error:
    
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	printf("Engine exited -- Enter to exit\n");
	scanf("exit");

	return cudaStatus;
}
