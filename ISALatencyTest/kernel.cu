#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "compute_capability.h"

#include <stdio.h>

typedef unsigned int(*TEST)(bool);

__device__ void Evaluate(TEST test)
{
	unsigned repetitionsleft = 5;
	unsigned lastlatency;
	unsigned newlatency;

	lastlatency = test(false);
	while (repetitionsleft) {
		newlatency = test(false);
		if (newlatency == lastlatency) repetitionsleft--;
		else {
			repetitionsleft = 5;
			lastlatency = newlatency;
		}
	}
	printf("%06u cycles: ", lastlatency);
	test(true);
	printf("\n");
}

#define NEXT_TEST_GROUP FirstTest

#include "test_bfind.h"
#include "test_brev.h"
#include "test_popc.h"
#include "test_template.h"											// Add your test_INSTRUCTION.h file here

__device__ void NEXT_TEST_GROUP() {}

__global__ void LatencyTest()
{
	FirstTest();
}

int main()
{
	hipError_t cudaStatus = hipSuccess;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

	LatencyTest <<<1, 1>>>();

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

Error:
    
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	printf("Engine exited -- Enter to exit\n");
	scanf("exit");

	return cudaStatus;
}
